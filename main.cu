#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <cmath>
#include <hip/hip_runtime.h>

// CUDA kernel to calculate sequence identity on GPU
__global__ void calculate_sequence_identity_gpu(const char* seq1, const char* seq2, float* result, size_t seq_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < seq_len) {
        result[idx] = (seq1[idx] == seq2[idx]) ? 1.0f : 0.0f;
    }
}

// Function to calculate sequence identity using GPU
float calculate_sequence_identity_cuda(const char* seq1, const char* seq2, size_t seq_len) {
    const int threads_per_block = 256;
    const int blocks = (seq_len + threads_per_block - 1) / threads_per_block;

    float* d_result;
    hipMalloc((void**)&d_result, seq_len * sizeof(float));
    calculate_sequence_identity_gpu<<<blocks, threads_per_block>>>(seq1, seq2, d_result, seq_len);

    float* h_result = new float[seq_len];
    hipMemcpy(h_result, d_result, seq_len * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (size_t i = 0; i < seq_len; ++i) {
        sum += h_result[i];
    }

    delete[] h_result;
    hipFree(d_result);

    return (100.0f * sum) / (seq_len / 2.0f);
}

// Function to check sequence validity
bool check_sequence_validity_cuda(const char* seq1, const std::vector<std::string>& dataset2_sequences, float threshold) {
    for (const auto& seq2 : dataset2_sequences) {
        float score = calculate_sequence_identity_cuda(seq1, seq2.c_str(), std::min(seq1.length(), seq2.length()));
        if (score > threshold) {
            return false;
        }
    }
    return true;
}

// Function to process a sequence
bool process_sequence_cuda(int index, const std::string& row, const std::vector<std::string>& dataset2_sequences, float threshold) {
    return check_sequence_validity_cuda(row.c_str(), dataset2_sequences, threshold);
}

int main() {
    // Load protein sequences from CSV files
    std::vector<std::string> dataset1_sequences = load_sequences_from_csv("/content/data_creation/Dataset/deeploc_data.csv");
    std::vector<std::string> dataset2_sequences = load_sequences_from_csv("/content/data_creation/Dataset/hpa_testset.csv");

    float threshold = 30.0;  // Set your desired threshold

    std::vector<bool> results(dataset1_sequences.size(), false);

    // Process sequences in parallel using threads
    #pragma omp parallel for
    for (size_t i = 0; i < dataset1_sequences.size(); ++i) {
        results[i] = process_sequence_cuda(i, dataset1_sequences[i], dataset2_sequences, threshold);
    }

    // Print results
    for (size_t i = 0; i < dataset1_sequences.size(); ++i) {
        std::cout << "Sequence " << i << ": " << (results[i] ? "Valid" : "Not Valid") << std::endl;
    }

    return 0;
}
